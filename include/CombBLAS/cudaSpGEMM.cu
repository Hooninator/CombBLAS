#include "hip/hip_runtime.h"


#include "cudaSpGEMM.h"
#include <cstdint>
#include <>
#include <stdio.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "../GALATIC/include/CSR.cuh"
#include "../GALATIC/include/dCSR.cuh"


template <typename NTO, typename IT, typename NT1, typename NT2>
__global__ void transformColumn_d(IT A_nzc, IT* A_Tran_CP,
    IT* A_Tran_IR,
    IT* A_Tran_JC,
    NT1* A_Tran_numx,
    IT* B_CP,
    IT* B_IR,
    IT* B_JC,
    NT2* B_numx,
    std::tuple<IT,IT,NTO> * tuplesC, IT* curptrC, IT B_nzc) {
        for(size_t i = blockIdx.x; i < B_nzc; i += gridDim.x) {
            size_t nnzcolB = B_CP[i+1] - B_CP[i];
                for(size_t j = threadIdx.x; j < A_nzc; j += blockDim.x) {
                bool made = false;
                size_t r = A_Tran_CP[j];
                uint ptr = curptrC[i];
                for (size_t k = 0; k < nnzcolB; ++k) {
                    
                    while (r < A_Tran_CP[j + 1] && B_IR[B_CP[i]+k] > A_Tran_IR[r]) { 
                        r++;
                    }
                    if (r >= A_Tran_CP[j + 1]) {
                            break;
                        }
                    if (B_IR[B_CP[i]+k] == A_Tran_IR[r]) {
                        NTO mrhs = A_Tran_numx[r] * B_numx[B_CP[i]+k];
                        if(true) {
                            if (made) {
                                std::get<2>(tuplesC[ptr]) = std::get<2>(tuplesC[ptr]) + mrhs;
                            } else {
                                made = true;
                                ptr = atomicAdd((unsigned long long*) &curptrC[i],(unsigned long long) 1);
                                std::get<0>(tuplesC[ptr]) = A_Tran_JC[j];
                                std::get<1>(tuplesC[ptr])= B_JC[i];
                                std::get<2>(tuplesC[ptr])  = mrhs;
                            }
                        }
                    }
                }
            }
        }
}
template < typename NTO, typename IT, typename NT1, typename NT2>
void transformColumn(IT A_nzc, IT* A_Tran_CP,
    IT* A_Tran_IR,
    IT* A_Tran_JC,
    NT1* A_Tran_numx,
    IT* B_CP,
    IT* B_IR,
    IT* B_JC,
    NT2* B_numx,
     std::tuple<IT,IT,NTO> * tuplesC_d, IT* curptrC, IT B_nzc) {
        int blks = std::min(65535,(int) B_nzc);
        transformColumn_d<<<blks,256>>>(A_nzc, A_Tran_CP,
    A_Tran_IR,
    A_Tran_JC,
     A_Tran_numx,
    B_CP,
B_IR,
    B_JC,
     B_numx,
    tuplesC_d, curptrC, B_nzc);
}

template void transformColumn< double, int64_t, double, double>(
   int64_t A_nzc, int64_t* A_Tran_CP,
    int64_t* A_Tran_IR,
    int64_t* A_Tran_JC,
    double* A_Tran_numx,
    int64_t* B_CP,
    int64_t* B_IR,
    int64_t* B_JC,
    double* B_numx,
    std::tuple<int64_t,int64_t,double> * tuplesC_d, int64_t* curptrC, int64_t B_nzc);

template <typename Arith_SR, typename NTO, typename NT1, typename NT2, typename IT>
__host__  CSR<NTO> LocalGalaticSPGEMM
(CSR<NT1> input_A_CPU,
CSR<NT2> input_B_CPU,
 bool clearA, bool clearB, Arith_SR semiring, IT * aux = nullptr) {
 }

template CSR<double> LocalGalaticSPGEMM<Arith_SR, double, double, double, int64_t>
(CSR<double> input_A_CPU,
CSR<double> input_B_CPU,
 bool clearA, bool clearB, Arith_SR semiring, int64_t * aux = nullptr);
